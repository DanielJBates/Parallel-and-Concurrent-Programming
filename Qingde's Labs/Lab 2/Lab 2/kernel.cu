#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 32

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, int* a, int* b)
{
    //int blockSize = blockDim.x * blockDim.y * blockDim.z;

    //int i = threadIdx.x + blockIdx.x * blockSize; //multiple 1D
    //int i = threadIdx.x + threadIdx.y * blockDim.x; //one 2D
    int i = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y);
    
    c[i] = a[i] + b[i];
}

__global__ void matAddKernel(int C[N][N], int A[N][N], int B[N][N])
{
    int i = threadIdx.x;
    int j = threadIdx.y;

    C[i][j] = A[i][j] + B[i][j];
}

void add(int* c, const int* a, const int* b)
{
    for (int i = 0; i < 5; i++)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    const int arraySize = 5;
    int a[arraySize] = {1,2,3,4,5};
    int b[arraySize] = {10,20,30,40,50};
    int c[arraySize] = { 0 };

    int A[N][N];
    int B[N][N];
    int C[N][N];

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i][j] = i + j;
            B[i][j] = (i + j) * 10;
        }
    }

    //for (int i = 0; i < arraySize; i++)
    //{
    //    a[i] = (i + 1);
    //    b[i] = ((i + 1) * 10);
    //}

    //hipEvent_t start, stop;

    //hipEventCreate(&start);
    //hipEventCreate(&stop);

    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //add(c, a, b);

    hipError_t cudaStatus;

    //int* dev_a = 0;
    //int* dev_b = 0;
    //int* dev_c = 0;

    int (*dA)[N];
    int (*dB)[N];
    int (*dC)[N];

    //cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    //if (cudaStatus != hipSuccess)
    //{
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    cudaStatus = hipMalloc((void**)&dA, (N * N) * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    //if (cudaStatus != hipSuccess)
    //{
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    cudaStatus = hipMalloc((void**)&dB, (N * N) * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //cudaStatus = hipMalloc((void**)&dev_c, arraySize * sizeof(int));
    //if (cudaStatus != hipSuccess)
    //{
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    cudaStatus = hipMalloc((void**)&dC, (N * N) * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //cudaStatus = hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess)
    //{
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    cudaStatus = hipMemcpy(dA, A, (N * N) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //cudaStatus = hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess)
    //{
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    cudaStatus = hipMemcpy(dB, B, (N * N) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //cudaStatus = hipMemcpy(dev_c, c, arraySize * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess)
    //{
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    cudaStatus = hipMemcpy(dC, C, (N * N) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //hipEventRecord(start, 0);
    //addKernel << <dim3(2,2), dim3(2,3)>> > (dev_c, dev_a, dev_b);
    matAddKernel << <1, dim3(32, 32) >> > (dC, dA, dB);
    //hipEventRecord(stop, 0);

    //hipEventSynchronize(stop);
    //float elapsedTime;
    //hipEventElapsedTime(&elapsedTime, start, stop);

    //printf("Time elapsed the execution of kernal %fn", elapsedTime);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize failed!");
        goto Error;
    }

    //cudaStatus = hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess)
    //{
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    cudaStatus = hipMemcpy(C, dC, (N * N) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    //hipFree(dev_c);
    //hipFree(dev_a);
    //hipFree(dev_b);

    hipFree(dC);
    hipFree(dA);
    hipFree(dB);

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    c[0], c[1], c[2], c[3], c[4]);

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}