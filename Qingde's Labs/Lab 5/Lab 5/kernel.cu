#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define rowsA 512
#define columnsA 512
#define rowsB 512
#define columnsB 512

//__shared__ int d_A[rowsA][columnsA], int d_B[rowsB][columnsB], int d_C[rowsA][columnsB];

__global__ void kernalMatMultipy(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    int j = threadIdx.y;

    int C_ij = i * blockDim.x + j;

    int temp = 0;
    for (int k = 0; k < rowsA; k++)
    {
        int i_A = i * columnsA + k;
        int i_B = k * columnsB + j;

        temp += a[i_A] * b[i_B];
    }

    c[C_ij] = temp;
}

void matMultiply(int a[rowsA][columnsA], int b[rowsB][columnsB], int c[rowsA][columnsB])
{
    for (int i = 0; i < rowsA; i++)
    {
        for (int j = 0; j < columnsB; j++)
        {
            for (int k = 0; k < rowsB; k++)
            {
                c[i][j] += a[i][k] * b[k][j];
            }
        }
    }
}

int main()
{
    int A[rowsA][columnsA];
    int B[rowsB][columnsB];
    int C[rowsA][columnsB] = { 0 };

    int x = 1;

    for (int i = 0; i < rowsA; i++)
    {
        for (int j = 0; j < columnsA; j++)
        {
            A[i][j] = x;
            x++;
        }
    }

    for (int i = 0; i < rowsB; i++)
    {
        for (int j = 0; j < columnsB; j++)
        {
            B[i][j] = x;
            x++;
        }
    }
    const int arraySizeA = rowsA * columnsA;
    const int arraySizeB = rowsB * columnsB;
    const int arraySizeC = rowsA * columnsB;

    int a[arraySizeA];

    x = 1;

    for (int i = 0; i < arraySizeA; i++)
    {
        a[i] = x;
        x++;
    }

    int b[arraySizeB];

    for (int i = 0; i < arraySizeB; i++)
    {
        b[i] = x;
        x++;
    }
    
    int c[arraySizeC] = { 0 };

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    int* d_A;
    int* d_B;
    int* d_C;

    hipMalloc((void**)&d_A, arraySizeA * sizeof(int));
    hipMalloc((void**)&d_B, arraySizeB * sizeof(int)); 
    hipMalloc((void**)&d_C, arraySizeC * sizeof(int));

    hipMemcpy(d_A, a, arraySizeA * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, arraySizeB * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, c, arraySizeC * sizeof(int), hipMemcpyHostToDevice);
    
    dim3 gridShape(1);
    dim3 blockShape(columnsA, rowsB);

    hipEventRecord(start, 0); 
    //matMultiply(A, B, C);
    kernalMatMultipy << <gridShape, blockShape>> > (d_C, d_A, d_B);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);

    printf("Time: %fn", time);

    hipDeviceSynchronize();

    hipMemcpy(c, d_C, arraySizeC * sizeof(int), hipMemcpyDeviceToHost);

    return 0;
}