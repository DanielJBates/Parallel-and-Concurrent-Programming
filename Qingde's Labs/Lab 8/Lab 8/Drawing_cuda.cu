#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable.h"
#include "hitable_list.h"

#include <hip/hip_vector_types.h>

 // includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;

//__device__ static int ticks = 1;
__device__ int timestep = 0.1;
static vec3 position = vec3(-1.5, 0, 0);
__device__ vec3 v = vec3(0.1, 0, 0);


hipArray* d_imageArray = 0;

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);

	}
}

__device__ vec3 castRay(const ray& r, hitable** world) {
	hit_record rec;
	if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
		vec3 colour = 0.5f * vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
		colour = 0.5 * rec.colour + 0.5 * colour;
		return colour;
	}
	else {
		vec3 unit_direction = unit_vector(r.direction());
		float t = 0.5f * (unit_direction.y() + 1.0f);
		return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
	}
}

__global__ void create_world(hitable** d_list, hitable** d_world) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		
		//vec3 v = vec3(0.01, 0.0, 0);


		//if (v.x() >= 2.0 || v.x() <= -2.0)
		//{
		//	position += vec3(v.x() * timestep,0,0);
		//	*(d_list) = new sphere(vec3(position.x() - v.x(), 0 + v.y(), -0.5 + v.z()), 0.5);
		//	
		//	//v = vec3(-0.01 * (float)ticks++, 0.0 * (float)ticks++, 0);
		//}

		//if (sqrt(((-10002.0 - (-1.5 + v.x())) * (-10002.0 - (-1.5 + v.x()))) + ((0 - (0 + v.y())) * (0 - (0 + v.y())))) <= 10000 + 0.5
		//	||
		//	sqrt(((10002.0 - (-1.5 + v.x())) * (10002.0 - (-1.5 + v.x()))) + ((0 - (0 + v.y())) * (0 - (0 + v.y())))) <= 10000 + 0.5) //refelect v on x axis
		//{
		//	v = vec3(-(v.x()), v.y(), v.z());
		//}

		//if (sqrt(((0 - (-1.5 + v.x())) * (0 - (-1.5 + v.x()))) + ((10002.0 - (0 + v.y())) * (10002.0 - (0 + v.y())))) <= 10000 + 0.5
		//	||
		//	sqrt(((0 - (-1.5 + v.x())) * (0 - (-1.5 + v.x()))) + ((-10002.0 - (0 + v.y())) * (-10002.0 - (0 + v.y())))) <= 10000 + 0.5) //refelect v on y axis
		//{
		//	v = vec3(v.x(), -(v.y()), v.z());
		//}


		*(d_list) = new sphere(vec3(position.x() + v.x(), position.y() + v.y(), position.z() + v.z()), 0.5);
		//*(d_list) = new sphere(vec3(cos(0.01 * (float)ticks++), sin(0.01 * (float)ticks++), -0.5), 0.5);
		*(d_list + 1) = new sphere(vec3(-10002.0, 0, -3), 10000); //left
		*(d_list + 2) = new sphere(vec3(10002.0, 0, -3), 10000); //right
		*(d_list + 3) = new sphere(vec3(0, 10002.0, -3), 10000); //top
		*(d_list + 4) = new sphere(vec3(0, -10002.0, -3), 10000); //bottom
		*(d_list + 5) = new sphere(vec3(0, 0, -10000.5), 10000); //back

		*d_world = new hitable_list(d_list, 6);
	}
}
__global__ void free_world(hitable** d_list, hitable** d_world) {
	delete* (d_list);
	delete* (d_list + 1);
	delete* (d_list + 2); 
	delete* (d_list + 3); 
	delete* (d_list + 4);
	delete* (d_list + 5);
	delete* d_world;
}

__global__ void d_render(uchar4* d_output, uint width, uint height, hitable** d_world)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint i = y * width + x;

	float u = x / (float)width; //----> [0, 1]x[0, 1]
	float v = y / (float)height;

	u = 2.0 * u - 1.0; //---> [-1, 1]x[-1, 1]
	v = -(2.0 * v - 1.0);
	u *= width / (float)height;

	u *= 2.0;
	v *= 2.0;

	vec3 eye = vec3(0, 0.5, 1.5);
	float distFrEye2Img = 1.0;;
	if ((x < width) && (y < height))
	{
		//for each pixel
		vec3 pixelPos = vec3(u, v, eye.z() - distFrEye2Img);
		//fire a ray:
		ray r;
		r.Origin = eye;
		r.Direction = pixelPos - eye; //view direction along negtive z-axis!
		vec3 col = castRay(r, d_world);
		float red = col.x();
		float green = col.y();
		float blue = col.z();
		d_output[i] = make_uchar4(red * 255, green * 255, blue * 255, 0);
	}
}


	extern "C" void freeTexture() {

		checkCudaErrors(hipFreeArray(d_imageArray));
	}

	// render image using CUDA
	extern "C" void render(int width, int height, dim3 blockSize, dim3 gridSize, uchar4 * output)
	{
		// make our world of hitables
		hitable** d_list;
		checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(hitable*)));
		hitable** d_world;
		checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
		create_world << <1, 1 >> > (d_list, d_world);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		d_render << <gridSize, blockSize >> > (output, width, height, d_world);
		getLastCudaError("kernel failed");
	}
#endif