#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void dotKernel(int *c, int *a, int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}

void dot(int* c, int* a, int* b, const int size)
{
    for (int i = 0; i < size; i++)
    {
        c[i] = a[i] * b[i];
    }
}

void sumOfDot(int *c, int &sum, const int size)
{
    for (int i = 0; i < size; i++)
    {
        sum += c[i];
    }
}

int main()
{
    const int arraySize = 5;
    int a[arraySize] = { 1, 2, 3, 4, 5 };
    int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    int sum = 0;

    hipError_t cudaStatus;

    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_c, c, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    dotKernel << <1, arraySize >> > (dev_c, dev_a, dev_b);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    sumOfDot(c, sum, arraySize);

    printf("{1,2,3,4,5} . {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);
    printf("\n");
    printf("Sum of {%d,%d,%d,%d,%d} = %d\n", c[0], c[1], c[2], c[3], c[4], sum);

    return 0;
}