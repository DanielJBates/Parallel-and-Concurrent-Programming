#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>

 // includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;


hipArray* d_imageArray = 0;

hipTextureObject_t rgbaTexdImage;


__global__ void d_render(uchar4* d_output, uint width, uint height, float tx,
    float ty, float scale, float cx, float cy,
    hipTextureObject_t texObj) {
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = __umul24(y, width) + x;
    
    //float2 T = { 200, 100 };
    //x += T.x;
    //y += T.y;

    //float2 S = { 1.2, 0.5 };
    //x *= S.x;
    //y *= S.y;

    //float angle = 0.5;
    //float rx = x * cos(angle) - y * sin(angle);
    //float ry = x * sin(angle) + y * cos(angle);
    
    //float2 S = { 1.2, 0.5 };
    //float u = (x - cx) * S.x + cx;
    //float v = (y - cy) * S.y + cy;
    
    //float x0 = width / 2.0;
    //float y0 = height / 2.0;

    //float angle = 0.5;
    //float rx = (x - x0) * cos(angle) - (y - y0) * sin(angle);
    //float ry = (x - x0) * sin(angle) + (y - y0) * cos(angle);

    //rx += x0;
    //ry += y0;

    //float u = (x - cx) * scale + cx + tx;
    //float v = (y - cy) * scale + cy + ty;

    if ((x < width) && (y < height)) {
        // write output color
        //float c = tex2D<float>(texObj, rx, ry);
        float centre = tex2D<float>(texObj, x, y);
        float left = tex2D<float>(texObj, x - 1, y);
        float right = tex2D<float>(texObj, x + 1, y);
        float up = tex2D<float>(texObj, x, y + 1);
        float down = tex2D<float>(texObj, x, y - 1);

        float c = (centre + left + right + up + down) / 5;

        d_output[i] = make_uchar4(c * 0xff, c * 0xff, c * 0xff, 0);
        //d_output[i] = make_uchar4(0xff, 0, 0, 0);
        //d_output[i] = make_uchar4(0, 0xff, 0, 0);
        //d_output[i] = make_uchar4(0, 0, c * 0xff, 0);
    }
}


extern "C" void initTexture(int imageWidth, int imageHeight, uchar * h_data) {
    // allocate array and copy image data
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors(
        hipMallocArray(&d_imageArray, &channelDesc, imageWidth, imageHeight));
    checkCudaErrors(hipMemcpy2DToArray(
        d_imageArray, 0, 0, h_data, imageWidth * sizeof(uchar),
        imageWidth * sizeof(uchar), imageHeight, hipMemcpyHostToDevice));
    free(h_data);

    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_imageArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(
        hipCreateTextureObject(&rgbaTexdImage, &texRes, &texDescr, NULL));

}

extern "C" void freeTexture() {

    checkCudaErrors(hipFreeArray(d_imageArray));
}

// render image using CUDA
extern "C" void render(int width, int height,  dim3 blockSize, dim3 gridSize,
     uchar4 * output) {

    float tx = 0, ty = 0, scale = 1, cx = 0, cy = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

        d_render << <gridSize, blockSize >> > (output, width, height, 0, 0, 1,
            0, 0, rgbaTexdImage);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);

    printf("Time: %fn", time);


    getLastCudaError("kernel failed");
}

#endif