#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void dotKernel(int* c, int* a, int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}

int main()
{
    const int arraySize = 5;
    int a[arraySize] = { 1, 2, 3, 4, 5 };
    int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    int sum = 0;

    hipError_t cudaStatus;

    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    cudaStatus = hipMallocManaged((void**)&dev_a, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMallocManaged((void**)&dev_b, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMallocManaged((void**)&dev_c, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_c, c, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    dotKernel << <1, arraySize >> > (dev_c, dev_a, dev_b);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    printf("{1,2,3,4,5} . {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    return 0;
}